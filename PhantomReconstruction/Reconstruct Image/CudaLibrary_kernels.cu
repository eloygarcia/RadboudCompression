#include "hip/hip_runtime.h"



__device__ void computealpha( const int  ind, const float origen, const float voxelSize,const float p1, const float p2, float &alpha) {	// alpha = [ X_plane(i) - X1] / [X2-X1] ;
		alpha = ((ind*voxelSize + origen)-p1)/(p2-p1); };

__device__ void Determinante( float* a0, float* a1, float* a2, float & det )
{
	float w[3] = {0.0,0.0,0.0};
		w[0] = a1[1]*a2[2] - a1[2]*a2[1];
		w[1] = -a1[0]*a2[2] + a1[2]*a2[0];
		w[2] = a1[0]*a2[1] - a1[1]*a2[0];
	
	det = a0[0]*w[0] + a0[1]*w[1] + a0[2]*w[2];
	//return det;
}

__device__ void triLinearInterpolator(const float* imagepointer, const int* size, const float* pixel, float & value) // es un pixel continuo, con lo cual, vale
{
//	float value = 0.0;
	int x0, y0, z0, x1, y1, z1;
		x0 = (int) (floorf(pixel[0])); x1=x0+1; // x1 = (int) (ceilf(pixel[0]));
		y0 = (int) (floorf(pixel[1])); y1=y0+1; // y1 = (int) (ceilf(pixel[1]));
		z0 = (int) (floorf(pixel[2])); z1=z0+1; // z1 = (int) (ceilf(pixel[2]));

	float xd = (pixel[0]-x0)/(x1-x0);
	float yd = (pixel[1]-y0)/(y1-y0);
	float zd = (pixel[2]-z0)/(z1-z0);

	int idx_1 = 0;		int idx_2 = 0;

		idx_1 = x0 + (size[0] * y0) + (size[0]*size[1]*z0);
		idx_2 = x1 + (size[0] * y0) + (size[0]*size[1]*z0);
	float c00 = ( imagepointer[idx_1] * (1-xd) ) + ( imagepointer[ idx_2 ] * (xd) );
		idx_1 = x0 + (size[0] * y0) + (size[0]*size[1]*z1);
		idx_2 = x1 + (size[0] * y0) + (size[0]*size[1]*z1);
	float c01 = ( imagepointer[idx_1] * (1-xd) ) + ( imagepointer[ idx_2 ] * (xd) );
		idx_1 = x0 + (size[0] * y1) + (size[0]*size[1]*z0);
		idx_2 = x1 + (size[0] * y1) + (size[0]*size[1]*z0);
	float c10 = ( imagepointer[idx_1] * (1-xd) ) + ( imagepointer[ idx_2 ] * (xd) );
		idx_1 = x0 + (size[0] * y1) + (size[0]*size[1]*z1);
		idx_2 = x1 + (size[0] * y1) + (size[0]*size[1]*z1);
	float c11 = ( imagepointer[idx_1] * (1-xd) ) + ( imagepointer[ idx_2 ] * (xd) );

		float c0 = c00 * (1-yd) + c10 * yd;
		float c1 = c01 * (1-yd) + c11 * yd;

		float c = c0 * (1-zd) + c1 * zd;

		value = c;
}

__device__ void computeBaricentricCoordinates(float* posicionPunto, float* vertex_0, float* vertex_1, float* vertex_2, float* vertex_3, float* baricentricCoordinates, bool & is_inside)
{
	is_inside = false;

	float vap[3] = {0.0,0.0,0.0};
		vap[0] = posicionPunto[0]-vertex_0[0];
		vap[1] = posicionPunto[1]-vertex_0[1];
		vap[2] = posicionPunto[2]-vertex_0[2];

	float vbp[3] = {0.0,0.0,0.0};
		vbp[0] = posicionPunto[0]-vertex_1[0];
		vbp[1] = posicionPunto[1]-vertex_1[1];
		vbp[2] = posicionPunto[2]-vertex_1[2];

	float vab[3] = {0.0,0.0,0.0};
		vab[0] = vertex_1[0]-vertex_0[0];
		vab[1] = vertex_1[1]-vertex_0[1];
		vab[2] = vertex_1[2]-vertex_0[2];

	float vac[3] = {0.0,0.0,0.0};
		vac[0] = vertex_2[0]-vertex_0[0];
		vac[1] = vertex_2[1]-vertex_0[1];
		vac[2] = vertex_2[2]-vertex_0[2];
	
	float vad[3] = {0.0,0.0,0.0};
		vad[0] = vertex_3[0]-vertex_0[0];
		vad[1] = vertex_3[1]-vertex_0[1];
		vad[2] = vertex_3[2]-vertex_0[2];

	float vbc[3] = {0.0,0.0,0.0};
		vbc[0] = vertex_2[0]-vertex_1[0];
		vbc[1] = vertex_2[1]-vertex_1[1];
		vbc[2] = vertex_2[2]-vertex_1[2];

	float vbd[3] = {0.0,0.0,0.0};
		vbd[0] = vertex_3[0]-vertex_1[0];
		vbd[1] = vertex_3[1]-vertex_1[1];
		vbd[2] = vertex_3[2]-vertex_1[2];

	float va6 = 0;
        Determinante(vbp, vbd, vbc, va6);
	float vb6 = 0;
        Determinante(vap, vac, vad, vb6);
	float vc6 = 0;
        Determinante(vap, vad, vab, vc6);
	float vd6 = 0;
        Determinante(vap, vab, vac, vd6);
    float v_temp = 0;
        Determinante(vab, vac, vad, v_temp);
	float v6 = 1/abs(v_temp);

	baricentricCoordinates[0] = va6*v6;
	baricentricCoordinates[1] = vb6*v6;
	baricentricCoordinates[2] = vc6*v6;
	baricentricCoordinates[3] = vd6*v6;

	
	if( (baricentricCoordinates[0]>=0 && baricentricCoordinates[0] <= 1) &&
		(baricentricCoordinates[1]>=0 && baricentricCoordinates[1] <= 1) &&
		(baricentricCoordinates[2]>=0 && baricentricCoordinates[2] <= 1) &&
		(baricentricCoordinates[3]>=0 && baricentricCoordinates[3] <= 1) &&
		baricentricCoordinates[0]+baricentricCoordinates[1]+baricentricCoordinates[2]+baricentricCoordinates[3]<1.01 &&
		baricentricCoordinates[0]+baricentricCoordinates[1]+baricentricCoordinates[2]+baricentricCoordinates[3]>0.99 )
		is_inside=true;
	
	//return is_inside;
}

__device__ void computeCartessianCoordinates(float* baricentricCoordinates, float* vertex_0, float* vertex_1, float* vertex_2, float* vertex_3, float* posicionPunto)
{
	posicionPunto[0] = baricentricCoordinates[0]*vertex_0[0] + baricentricCoordinates[1]*vertex_1[0] + baricentricCoordinates[2]*vertex_2[0] + baricentricCoordinates[3]*vertex_3[0];
	posicionPunto[1] = baricentricCoordinates[0]*vertex_0[1] + baricentricCoordinates[1]*vertex_1[1] + baricentricCoordinates[2]*vertex_2[1] + baricentricCoordinates[3]*vertex_3[1];
	posicionPunto[2] = baricentricCoordinates[0]*vertex_0[2] + baricentricCoordinates[1]*vertex_1[2] + baricentricCoordinates[2]*vertex_2[2] + baricentricCoordinates[3]*vertex_3[2];
}

/*
__global__ void kernel_projection(const int* dev_3d_size, const float* dev_3d_spacing, const float* dev_3d_origen, const float* dev_3d_imagepointer,
								  const int* dev_2d_size, const float* dev_2d_spacing, const float* dev_2d_origen, float* dev_2d_imagepointer,
								  const float* source)
{
	int i = (blockDim.x * blockIdx.x) + threadIdx.x;
	
	int numberOfPixels2d = dev_2d_size[0] * dev_2d_size[1]; // *size2d[2];

	if( i<numberOfPixels2d){

	// Fila y columna de la imagen !
	int row = (int) floorf( i / dev_2d_size[0] ); 
	int col = (int) ( i - (row*dev_2d_size[0]) ); 
	

	// Posición del pixel del detector !!
	float x2 = dev_2d_origen[0] + (col * dev_2d_spacing[0] );
	float y2 = dev_2d_origen[1] + (row * dev_2d_spacing[1] );

	// Vector de dirección !!
	float vect[3];
		vect[0] = x2 - source[0];
		vect[1] = y2 - source[1];
		vect[2] = dev_2d_origen[2] - source[2];  // Revisar este punto.

	// Distancia de la fuente al detector !!
	float xa = pow(vect[0],2);
	float ya = pow(vect[1],2);
	float za = pow(vect[2],2);

	float dist12 = sqrt( xa + ya + za );

	// Cálculo del alpha en Z !!
	float temp = 0.0f;
	computealpha( 0, dev_3d_origen[2], dev_3d_spacing[2], source[2], dev_2d_origen[2], temp );

	// Resolver la ecuación de la recta !!
	float temp_dist[3] = {0.0f, 0.0f, 0.0f};
	float point[3] = {0.0f, 0.0f, 0.0f};
	float step = 0.0005f;
	float t=temp;

	float pixel[3] = {0.0f, 0.0f, 0.0f};

	float value = 0.0f;
	float length = 0.0f;

	// Longitud del paso !!
	temp_dist[0] = pow(step*vect[0],2);
	temp_dist[1] = pow(step*vect[1],2);
	temp_dist[2] = pow(step*vect[2],2);
	float l_step = sqrt(temp_dist[0] + temp_dist[1] + temp_dist[2]);

	while( t<1 )
	{
		// Calculo del siguiente punto en la recta
		point[0] = source[0] + t * vect[0];
		point[1] = source[1] + t * vect[1];
		point[2] = source[2] + t * vect[2];
		// Posición del voxel parcial !!
		pixel[0] = (point[0] - dev_3d_origen[0]) / dev_3d_spacing[0];
		pixel[1] = (point[1] - dev_3d_origen[1]) / dev_3d_spacing[1];
		pixel[2] = (point[2] - dev_3d_origen[2]) / dev_3d_spacing[2];
		// Interpolación trilienal
		if((pixel[0]<0 || pixel[0]>dev_3d_size[0]-1) || (pixel[1]<0 || pixel[1]>dev_3d_size[1]-1) || (pixel[2]<0 || pixel[2]>dev_3d_size[2]-1))  value = 0;
		else triLinearInterpolator( dev_3d_imagepointer, dev_3d_size, pixel, value);
		// Longitud acumulada !!
		length += (1000 * value * l_step);
		// Nuevo punto!! 
		t+=step;
	}

	if(length>0 & length < 65535 ) dev_2d_imagepointer[ i ] = length;
	else dev_2d_imagepointer[ i ] = 0.0f;
	
	}
}
*/

__global__ void kernel_projection(const int* dev_3d_size, const float* dev_3d_spacing, const float* dev_3d_origen, const unsigned char* dev_3d_imagepointer,
								  const float* dev_i_points, const float* dev_f_points, const int* dev_elements,
								  const float* dev_grid_origen, const float* dev_grid_spacing, const int* dev_grid_size,
								  const int* dev_flags, const int* dev_cumsum, const int* dev_correspondingElements,
								  const int* dev_2d_size, const float* dev_2d_spacing, const float* dev_2d_origen, unsigned char* dev_2d_imagepointer) 
{
	long int i = (blockDim.x * blockIdx.x) + threadIdx.x;
	
	//int numberOfPixels2d = dev_2d_size[0] * dev_2d_size[1] * dev_2d_size[2];

		int index_z = (int)  (i /((long int)dev_2d_size[0]*(long int)dev_2d_size[1]));
		int index_y = (int) ((i - ((long int) index_z*(long int)dev_2d_size[0]*(long int)dev_2d_size[1]))/(long int) dev_2d_size[0]);
		int index_x = (int) ( i - ((long int) index_z*(long int)dev_2d_size[0]*(long int)dev_2d_size[1]) - ((long int) index_y*(long int)dev_2d_size[0]));

		float temp_position[3] = {0.0,0.0,0.0};
			temp_position[0] = dev_2d_origen[0] + dev_2d_spacing[0]*(index_x + 0.5);
			temp_position[1] = dev_2d_origen[1] + dev_2d_spacing[1]*(index_y + 0.5);
			temp_position[2] = dev_2d_origen[2] + dev_2d_spacing[2]*(index_z + 0.5);

		float temp_VoxelGrid[3] = {0.0,0.0,0.0};
			temp_VoxelGrid[0] = floorf((temp_position[0] - dev_grid_origen[0])/dev_grid_spacing[0]);
			temp_VoxelGrid[1] = floorf((temp_position[1] - dev_grid_origen[1])/dev_grid_spacing[1]);
			temp_VoxelGrid[2] = floorf((temp_position[2] - dev_grid_origen[2])/dev_grid_spacing[2]);
		long int correspondingIndex = 0;
			correspondingIndex = temp_VoxelGrid[0] + (dev_grid_size[0]*temp_VoxelGrid[1]) + (dev_grid_size[0]*dev_grid_size[1]*temp_VoxelGrid[2]);
			
		// El pixel le tengo. Puedo saber qué elementos hay
		int index_start = dev_cumsum[ correspondingIndex ]; // Este i no vale! "i" es para la imagen!" hay que calcular otro para la grid regular
		int number_of_elements_here = dev_flags[ correspondingIndex ];
		// int count=0;
		int element_index_number = 0;
		int element_number = 0;

		int temp_element[4] = {0,0,0,0};
		float temp_vertex_0[3] = {0.0,0.0,0.0};
		float temp_vertex_1[3] = {0.0,0.0,0.0};
		float temp_vertex_2[3] = {0.0,0.0,0.0};
		float temp_vertex_3[3] = {0.0,0.0,0.0};

		float old_vertex_0[3] = {0.0,0.0,0.0};
		float old_vertex_1[3] = {0.0,0.0,0.0};
		float old_vertex_2[3] = {0.0,0.0,0.0};
		float old_vertex_3[3] = {0.0,0.0,0.0};

		float baricentricCoordinates[4] = {0.0,0.0,0.0,0.0};
		bool is_inside = false;

		float pixel[3] ={0.0,0.0,0.0};
		float sss_position[3] = {0.0,0.0,0.0};

		long int index = 0;
		float pixel_value = 0;

		if( number_of_elements_here != 0 ){
		for( int j=0; j<number_of_elements_here; j++ ){
			element_index_number = index_start + j;
			element_number = dev_correspondingElements[ element_index_number ];
			
				temp_element[0] = dev_elements[ 4*element_number ];
				temp_element[1] = dev_elements[ 4*element_number +1 ];
				temp_element[2] = dev_elements[ 4*element_number +2 ];
				temp_element[3] = dev_elements[ 4*element_number +3 ];
		
				temp_vertex_0[0] = dev_f_points[ 3*temp_element[0]  ];
				temp_vertex_0[1] = dev_f_points[ 3*temp_element[0] +1 ];
				temp_vertex_0[2] = dev_f_points[ 3*temp_element[0] +2 ];

				temp_vertex_1[0] = dev_f_points[ 3*temp_element[1] ];
				temp_vertex_1[1] = dev_f_points[ 3*temp_element[1] +1 ];
				temp_vertex_1[2] = dev_f_points[ 3*temp_element[1] +2 ];
				
				temp_vertex_2[0] = dev_f_points[ 3*temp_element[2] ];
				temp_vertex_2[1] = dev_f_points[ 3*temp_element[2] +1 ];
				temp_vertex_2[2] = dev_f_points[ 3*temp_element[2] +2 ];
				
				temp_vertex_3[0] = dev_f_points[ 3*temp_element[3] ];
				temp_vertex_3[1] = dev_f_points[ 3*temp_element[3] +1 ];
				temp_vertex_3[2] = dev_f_points[ 3*temp_element[3] +2 ];

			computeBaricentricCoordinates(temp_position, temp_vertex_0, temp_vertex_1, temp_vertex_2, temp_vertex_3, baricentricCoordinates, is_inside);

			if( is_inside ){
				// dev_2d_imagepointer[i] = 0; // hasta aquí llega!!!
				//if(dev_2d_imagepointer[i] == 0 ) 
				//{
					// Primer código !
//					 final_image[i] = element_number+1;
					// Segundo código !
					// 1. Hay que acalcular primero la posición cartesiana en el espacio de la imagen de entrada
				//	std::cout << "Temp. Element: [" << temp_element[0] << ", " << temp_element[1] << ", " << temp_element[2] << ", " << temp_element[3] << "] " << std::endl;
					
						old_vertex_0[0] = dev_i_points[ 3*temp_element[0]  ];
						old_vertex_0[1] = dev_i_points[ 3*temp_element[0] +1 ];
						old_vertex_0[2] = dev_i_points[ 3*temp_element[0] +2 ];
		
						old_vertex_1[0] = dev_i_points[ 3*temp_element[1] ];
						old_vertex_1[1] = dev_i_points[ 3*temp_element[1] +1 ];
						old_vertex_1[2] = dev_i_points[ 3*temp_element[1] +2 ];
					
						old_vertex_2[0] = dev_i_points[ 3*temp_element[2] ];
						old_vertex_2[1] = dev_i_points[ 3*temp_element[2] +1 ];
						old_vertex_2[2] = dev_i_points[ 3*temp_element[2] +2 ];
									
						old_vertex_3[0] = dev_i_points[ 3*temp_element[3] ];
						old_vertex_3[1] = dev_i_points[ 3*temp_element[3] +1 ];
						old_vertex_3[2] = dev_i_points[ 3*temp_element[3] +2 ];

					computeCartessianCoordinates( baricentricCoordinates, old_vertex_0, old_vertex_1, old_vertex_2, old_vertex_3, sss_position);

						// 2. Localizamos el pixel continuo e interpolamos el valor. 
					
					//	pixel[0] = (sss_position[0]-origenCuda[0])/spacingCuda[0];
					//	pixel[1] = (sss_position[1]-origenCuda[1])/spacingCuda[1];
					//	pixel[2] = (sss_position[2]-origenCuda[2])/spacingCuda[2];
						

						pixel[0] = (sss_position[0]-dev_3d_origen[0])/dev_3d_spacing[0];
						pixel[1] = (sss_position[1]-dev_3d_origen[1])/dev_3d_spacing[1];
						pixel[2] = (sss_position[2]-dev_3d_origen[2])/dev_3d_spacing[2];

												//Checkeando la información ! 
//					std::cout << "Temporal position. [" << temp_position[0] <<", " << temp_position[1] << ", " << temp_position[2] << "] " << std::endl;
//					std::cout << "    coordenadas baricentricas: [" << baricentricCoordinates[0] << ", " << baricentricCoordinates[1] << ", " << baricentricCoordinates[2] << ", " << baricentricCoordinates[3] << "] " << std::endl;
//					std::cout << "Reconstructed position. [" << sss_position[0] <<", " << sss_position[1] << ", " << sss_position[2] << "] " << std::endl;
//					std::cout << " El pixel es [" << pixel[0] <<", " << pixel[1] << ", " << pixel[2] << "]  de size = [" << sizeCuda[0] <<", " << sizeCuda[1] << ", "<< sizeCuda[2] << "] " << std::endl;
//					std::cout << std::endl;
					// Sleep(250);

					index = (long int)((long int)floorf(pixel[0]) + ((long int)dev_3d_size[0] * (long int)floorf(pixel[1]) ) + ((long int)dev_3d_size[0]*(long int)dev_3d_size[1]*(long int)floorf(pixel[2])));
					//dev_2d_imagepointer[i] = (unsigned char)index;
					if(index>0) dev_2d_imagepointer[i]=dev_3d_imagepointer[index];

  					
					// dev_2d_imagepointer[i] = (unsigned char)dev_3d_imagepointer[index];
					 
					//triLinearInterpolator(dev_3d_imagepointer, dev_3d_size, pixel, pixel_value);
//						dev_2d_imagepointer[i] = pixel_value;
					//else std::cout << "Fuera de image!" << std::endl;


//						trilinearinterpolation( image_input , sizeCuda, pixel, value);
//						nearestNeighborInterpolation( image_input , sizeCuda, pixel, value);
						// Hay que cambiar esta interpolacion por nearest neigbors

//						final_image[i] = value;
				/*	*/

				//}else{
				//	count++;
				//	std::cout << i << ".  Más de un elemento en el voxel !! " << final_image[i] << " and " << element_number << std::endl;
				//	std::cout << "    coordenadas baricentricas: [" << barcor[0] << ", " << barcor[1] << ", " << barcor[2] << ", " << barcor[3] << "] " << std::endl;
				//	std::cout << "    coordenadas baricentricas: [" << baricentricCoordinates[0] << ", " << baricentricCoordinates[1] << ", " << baricentricCoordinates[2] << ", " << baricentricCoordinates[3] << "] " << std::endl;
				//	std::cout << std::endl;
				//	Sleep(500);

				// } // endif( final_image[i]==0 )
			} // endif( is_inside == true )		
		} // endfor
		} // endif( number_of_elements_here!=0 )		

		// dev_2d_imagepointer[i] = (char)number_of_elements_here;
		//dev_2d_imagepointer[i] = 1.0f;
}


__global__ void fill_dos( char * imagepointer)
{
	long unsigned int i = blockDim.x * blockIdx.x +threadIdx.x;
	imagepointer[i]=2;
}