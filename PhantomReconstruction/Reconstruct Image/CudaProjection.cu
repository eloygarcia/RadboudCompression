#include "hip/hip_runtime.h"
#include "CudaProjection.h"
#include "CudaLibrary_kernels.cu"

#include "hip/hip_runtime.h"
#include ""

// Constructor y Destructor

CudaProjection::CudaProjection()
{
	// initial ct image
	m_initial_size = new int[3];
	m_initial_spacing = new float[3];
	m_initial_origen = new float[3];
	numberOfPixels = 0;

    // grid
	m_grid_origen = new float[3];
	m_grid_spacing = new float[3];
	m_grid_size = new int[3];
	

	// mamo simulada
	m_simulada_size = new int[3];
	m_simulada_spacing = new float[3];
	m_simulada_origen = new float[3];
	numberOfPixels_Simulada=0;

	hipError_t cudaStatus;
}

CudaProjection::~CudaProjection()
{
	delete[] m_initial_size;
	delete[] m_initial_spacing;
	delete[] m_initial_origen;
	delete[] m_initial_imagepointer;

	delete [] m_i_points;
	delete [] m_f_points;
	delete [] m_elements;

	delete [] m_grid_origen;
	delete [] m_grid_size;
	delete [] m_grid_spacing;

	delete [] m_flags;
	delete [] m_cumsum;
	delete [] m_correspondingElements;

	delete[] m_simulada_size;
	delete[] m_simulada_spacing;
	delete[] m_simulada_origen;
}

void CudaProjection::Initialize()
{
	/*m_mri_size[0] = m_parameters->mri_size[0];
	m_mri_size[1] = m_parameters->mri_size[1];
	m_mri_size[2] = m_parameters->mri_size[2];*/
// numberOfPixels_MRI = m_mri_size[0] * m_mri_size[1] * m_mri_size[2];

	/*m_mri_spacing[0] = m_parameters->mri_spacing[0];
	m_mri_spacing[1] = m_parameters->mri_spacing[1];
	m_mri_spacing[2] = m_parameters->mri_spacing[2];*/

	/*m_mri_origen[0] = m_parameters->mri_origen[0];
	m_mri_origen[1] = m_parameters->mri_origen[1];
	m_mri_origen[2] = m_parameters->mri_origen[2];*/

//	m_mri_imagepointer = new float[ numberOfPixels_MRI ];
//		for(int i=0; i<numberOfPixels_MRI; i++) m_mri_imagepointer[i] = m_parameters->mri_imagePointer[i];

// numberOfPoints = m_parameters->numberOfPoints;
//	m_i_points = new float[ 3*numberOfPoints ];
//		for( int i=0; i<3*numberOfPoints; i++) m_i_points[i] = m_parameters->initial_points[i];
//	m_f_points = new float[ 3*numberOfPoints ];
//		for(int i=0; i<3*numberOfPoints; i++) m_f_points[i] = m_parameters->final_points[i];
// numberOfElements = m_parameters->numberOfElements;
//	m_elements = new int[4*numberOfElements];	
//	for(int i=0; i<4*numberOfElements; i++) m_elements[i] = m_parameters->elements[i];
//
//	m_grid_origen[0] = m_parameters->grid_origen[0];
//	m_grid_origen[1] = m_parameters->grid_origen[1];
//	m_grid_origen[2] = m_parameters->grid_origen[2];
//
//	m_grid_spacing[0] = m_parameters->grid_spacing[0];
//	m_grid_spacing[1] = m_parameters->grid_spacing[1];
//	m_grid_spacing[2] = m_parameters->grid_spacing[2];
//
//	m_grid_size[0] = m_parameters->grid_size[0];
//	m_grid_size[1] = m_parameters->grid_size[1];
//	m_grid_size[2] = m_parameters->grid_size[2];
// numberOfVoxelsGrid = m_grid_size[0] * m_grid_size[1] * m_grid_size[2] ; 

	//m_flags = new int[ numberOfVoxelsGrid ];
	//	for(int i=0; i<numberOfVoxelsGrid; i++) m_flags[i] = m_parameters->flags[i];
	//m_cumsum = new int[ numberOfVoxelsGrid ];
	//	for(int i=0; i<numberOfVoxelsGrid; i++) m_cumsum[i] = m_parameters->cumsum[i];
//maximumCorrespondingElements = m_cumsum[ numberOfVoxelsGrid -1];
//	m_correspondingElements = new int[ maximumCorrespondingElements];
//	for (int i=0; i< maximumCorrespondingElements; i++)	m_correspondingElements[i] = m_parameters->correspondingElements[i];
//
//	m_simulada_size[0] = m_parameters->mamo_size[0];
//	m_simulada_size[1] = m_parameters->mamo_size[1];
//	m_simulada_size[2] = m_parameters->mamo_size[2];
//numberOfPixels_Simulada = m_simulada_size[0] * m_simulada_size[1];
//
//	m_simulada_origen[0] = m_parameters->mamo_origen[0];
//	m_simulada_origen[1] = m_parameters->mamo_origen[1];
//	m_simulada_origen[2] = m_parameters->mamo_origen[2];
//
//	m_simulada_spacing[0] = m_parameters->mamo_spacing[0];
//	m_simulada_spacing[1] = m_parameters->mamo_spacing[1];
//	m_simulada_spacing[2] = m_parameters->mamo_spacing[2];
//	
//		//m_simulada_imagepointer = new float[numberOfPixels_Simulada];
//	m_simulada_imagepointer = new unsigned short[numberOfPixels_Simulada];
//
//	m_source[0] = m_parameters->source[0];
//	m_source[1] = m_parameters->source[1];
//	m_source[2] = m_parameters->source[2];
}


void CudaProjection::SetInitialImage(int* imageSize, 
                                    float* imageSpacing,
                                    float* imageOrigin, 
                                    unsigned char* imagePointer)
{
    m_initial_size = imageSize;
numberOfPixels = (long)imageSize[0]*(long)imageSize[1]*(long)imageSize[2];
    m_initial_spacing = imageSpacing;
    m_initial_origen = imageOrigin;
    m_initial_imagepointer = imagePointer;
}

void CudaProjection::SetMesh(int numPoints, float* i_points, float* f_points, int numElements, int* elements)
{
numberOfPoints = numPoints;
    m_i_points = i_points;
    m_f_points = f_points;
numberOfElements = numElements;
    m_elements = elements;
}

void CudaProjection::SetGrid(int* gridSize, float* gridSpacing, float* gridOrigin,
                            int* flags, int* cumsum, int* correspondingElements)
{
    m_grid_size = gridSize;
numberOfVoxelsGrid = gridSize[0]*gridSize[1]*gridSize[2];
    m_grid_spacing = gridSpacing;
    m_grid_origen = gridOrigin;

    m_flags = flags;
    m_cumsum = cumsum;
    m_correspondingElements = correspondingElements;
maximumCorrespondingElements = cumsum[numberOfVoxelsGrid -1];
}

void CudaProjection::SetFinalImage(int* finalSize, float* finalSpacing, float* finalOrigin, unsigned char* finalPointer)
{
    m_simulada_size = finalSize;
    m_simulada_spacing = finalSpacing;
    m_simulada_origen = finalOrigin;
numberOfPixels_Simulada = (long)finalSize[0]*(long)finalSize[1]*(long)finalSize[2];
    m_simulada_imagepointer = finalPointer;
}
// Metodos
void CudaProjection::Update()
{
	/*
	printf("\n");
	printf("Entra en cuda\n");
	printf("\n");

// Timer !
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);
	*/

// Initialize(); // === Supongo inicializado desde el principio.
	m_simulada_imagepointer = new unsigned char[numberOfPixels_Simulada];

// Allocacion de la memoria GPU !
	// MRI !
    cudaStatus = hipMalloc((void**)&dev_initial_size, 3*sizeof(int));
	    if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc initial Size!\n");   // MRI Size
	cudaStatus = hipMalloc((void**)&dev_initial_spacing, 3*sizeof(float));
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc initial Spacing!\n"); // MRI Spacing
	cudaStatus = hipMalloc((void**)&dev_initial_origen, 3*sizeof(float));
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc initial Origen!\n");  // MRI Origen

	cudaStatus = hipMalloc((void**)&dev_initial_imagepointer, numberOfPixels*sizeof(unsigned char));
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc initial image pointer!\n");  // MRI imagen !!

	// Mesh & Grid !!
	cudaStatus = hipMalloc((void**)&dev_i_points, 3*numberOfPoints*sizeof(float)); 
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA alloc i_point !\n"); // i_points
	cudaStatus = hipMalloc((void**)&dev_f_points, 3*numberOfPoints*sizeof(float));
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA alloc f_point !\n"); // f_points

	cudaStatus = hipMalloc((void**)&dev_elements, 4*numberOfElements*sizeof(int));
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA alloc elements !\n"); // elements;

	cudaStatus = hipMalloc((void**)&dev_grid_origen, 3*sizeof(float));
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA alloc Grid Origen !\n"); // grid_origen
	cudaStatus = hipMalloc((void**)&dev_grid_spacing, 3*sizeof(float));
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA alloc Grid spacing!\n"); // grid_Spacing
	cudaStatus = hipMalloc((void**)&dev_grid_size, 3*sizeof(int));
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA alloc Grid Size !\n"); // grid_Size

	cudaStatus = hipMalloc((void**)&dev_flags, numberOfVoxelsGrid*sizeof(int));
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA alloc Flags !\n"); // flags
	cudaStatus = hipMalloc((void**)&dev_cumsum, numberOfVoxelsGrid*sizeof(int));
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA alloc CumSum !\n"); // cumsum
	cudaStatus = hipMalloc((void**)&dev_correspondingElements, maximumCorrespondingElements*sizeof(int));
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA alloc corresponding element !\n"); // corresponding Elements !

	// Imagen Simulada !
	cudaStatus = hipMalloc((void**)&dev_simulada_size, 3*sizeof(int));
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc simulada Size!\n");  // Simulada Size
	cudaStatus = hipMalloc((void**)&dev_simulada_spacing, 3*sizeof(float));
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc simulada Spacing!\n");  // Simulada Spacing
	cudaStatus = hipMalloc((void**)&dev_simulada_origen, 3*sizeof(float));
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc simulada Origen!\n");  // simulada Origen

	cudaStatus = hipMalloc((void**)&dev_simulada_imagepointer, numberOfPixels_Simulada*sizeof(unsigned char)); // u
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc simulada image pointer!\n");  // Simulada Imagen !!
	cudaStatus = hipMemset((void*)dev_simulada_imagepointer, 0, numberOfPixels_Simulada*sizeof(unsigned char));
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc simulada image pointer!\n");  // Inicializaci\F3n de la imagen simulada a Zeros !!

	// Source !
//	cudaStatus = hipMalloc((void**)&dev_source, 3*sizeof(float));
//		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc Source!\n");  // source !"


// Copia a memoria device
	// mri
	cudaStatus = hipMemcpy(dev_initial_size, (const int*) m_initial_size, 3*sizeof(int), hipMemcpyHostToDevice);
	// cudaStatus = hipMemcpy(dev_mri_size, (const int*) m_mri_size, 3*sizeof(int), hipMemcpyHostToDevice);
	    if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy initial Size Hostsimuladaev!\n");
	cudaStatus = hipMemcpy(dev_initial_spacing, (const float*) m_initial_spacing, 3*sizeof(float), hipMemcpyHostToDevice);
	// cudaStatus = hipMemcpy(dev_mri_spacing, this->m_mri_spacing, 3*sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy initial Spacing Hostsimuladaev!\n");
	cudaStatus = hipMemcpy(dev_initial_origen, (const float*) m_initial_origen, 3*sizeof(float), hipMemcpyHostToDevice);
	// cudaStatus = hipMemcpy(dev_mri_origen, this->m_mri_origen, 3*sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy initial Origen Hostsimuladaev!\n");
	cudaStatus = hipMemcpy(dev_initial_imagepointer, (const unsigned char*) m_initial_imagepointer, numberOfPixels*sizeof(unsigned char), hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy(dev_mri_imagepointer, this->m_mri_imagepointer, numberOfPixels_MRI*sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy initial image pointer Hostsimuladaev!\n");

	// Mesh & Grid !!
	cudaStatus = hipMemcpy(dev_i_points, (const float*) m_i_points, 3*numberOfPoints*sizeof(float), hipMemcpyHostToDevice); 
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA memcopy i_point Hostsimuladaev !\n"); // i_points
	cudaStatus = hipMemcpy(dev_f_points, (const float*) m_f_points,  3*numberOfPoints*sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA memcopy f_point Hostsimuladaev !\n"); // f_points

	cudaStatus = hipMemcpy(dev_elements, (const int*) m_elements, 4*numberOfElements*sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA memcopy elements Hostsimuladaev !\n"); // elements;

	cudaStatus = hipMemcpy(dev_grid_origen, (const float*) m_grid_origen, 3*sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA memcopy Grid Origen !\n"); // grid_origen
	cudaStatus = hipMemcpy(dev_grid_spacing, (const float*) m_grid_spacing, 3*sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA memcopy Grid spacing!\n"); // grid_Spacing
	cudaStatus = hipMemcpy(dev_grid_size, (const int*) m_grid_size, 3*sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA memcopy Grid Size !\n"); // grid_Size

	cudaStatus = hipMemcpy(dev_flags, (const int*) m_flags, numberOfVoxelsGrid*sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA memcpy Flags !\n"); // flags
	cudaStatus = hipMemcpy(dev_cumsum, (const int*) m_cumsum, numberOfVoxelsGrid*sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA memcpy CumSum !\n"); // cumsum
	cudaStatus = hipMemcpy(dev_correspondingElements, (const int*) m_correspondingElements, maximumCorrespondingElements*sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA memcpy corresponding element !\n"); // corresponding Elements !

	// mamo simulada
	cudaStatus = hipMemcpy(dev_simulada_size, (const int*) m_simulada_size, 3*sizeof(int), hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy(dev_simulada_size, this->m_simulada_size, 3*sizeof(int), hipMemcpyHostToDevice);
	    if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy simulada Size Hostsimuladaev!\n");
	cudaStatus = hipMemcpy(dev_simulada_spacing, (const float*) m_simulada_spacing, 3*sizeof(float), hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy(dev_simulada_spacing, this->m_simulada_spacing, 3*sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy simulada Spacing Hostsimuladaev!\n");
	cudaStatus = hipMemcpy(dev_simulada_origen, (const float*) m_simulada_origen, 3*sizeof(float), hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy(dev_simulada_origen, this->m_simulada_origen, 3*sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy simulada Origen Hostsimuladaev!\n");
//	cudaStatus = hipMemcpy(dev_simulada_imagepointer, this->m_simulada_imagepointer, numberOfPixels_simulada*sizeof(float), hipMemcpyHostToDevice);
//		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy simulada image pointer Hostsimuladaev!\n");

//	cudaStatus = hipMemcpy(dev_source, (const float*) m_source, 3*sizeof(float), hipMemcpyHostToDevice);
//	//cudaStatus = hipMemcpy(dev_source, this->m_source, 3*sizeof(float), hipMemcpyHostToDevice);
//	    if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy initial Source Hostsimuladaev!\n");


		// Checkin memory !!
		printf("\n");
		printf("initial size: [ %i,%i,%i]\n", m_initial_size[0],m_initial_size[1],m_initial_size[2]);
		printf("initial Spacing : [ %f,%f,%f]\n",m_initial_spacing[0], m_initial_spacing[1], m_initial_spacing[2]);
		printf("initial Origen : [ %f,%f,%f]\n",m_initial_origen[0], m_initial_origen[1], m_initial_origen[2]);
		printf("Number of Pixels : %lu\n",numberOfPixels);
		
		printf("simulada Size : [ %d,%d,%d]\n",m_simulada_size[0], m_simulada_size[1], m_simulada_size[2]);
		printf("simulada Spacing : [ %f,%f,%f]\n",m_simulada_spacing[0], m_simulada_spacing[1], m_simulada_spacing[2]);
		printf("simulada Origen : [ %f,%f,%f]\n",m_simulada_origen[0], m_simulada_origen[1], m_simulada_origen[2]);
		printf("Number of Pixels : %lu\n",numberOfPixels_Simulada);
		
		//printf("Source : [ %f,%f,%f]\n",m_source[0], m_source[1], m_source[2]);
		printf("\n");


		// bl = (int)ceilf((float)(numberOfPixels_Simulada/512))+1;
		bl = (unsigned int) (1+(numberOfPixels_Simulada/((long)512)));
		printf("Number of blocks: %d\n", bl);
		printf("\n"); 
// Kernel de proyecci\F3n?
		printf("Entra en el kernel\n" );
		 // fill_dos <<< bl,512 >>> (dev_simulada_imagepointer);
		
	//cudaStatus = hipSetDevice(0);
	kernel_projection <<< bl,512 >>> (dev_initial_size, dev_initial_spacing, dev_initial_origen, dev_initial_imagepointer,
										dev_i_points, dev_f_points, dev_elements,
										dev_grid_origen, dev_grid_spacing, dev_grid_size,
										dev_flags, dev_cumsum, dev_correspondingElements,
									  dev_simulada_size, dev_simulada_spacing, dev_simulada_origen, dev_simulada_imagepointer);
									  //dev_source);								   
					   
	hipDeviceSynchronize();

	hipError_t error = hipGetLastError();
	if(error!=hipSuccess)
	{
   		fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
   		exit(-1);
	}
	printf("Sale del kernel\n" );		

// Copia a memoria host
/*	int temp_initialsize[3] = {0,0,0};
	cudaStatus = hipMemcpy( temp_initialsize, (const int*) dev_initial_size, 3*sizeof(int),  hipMemcpyDeviceToHost);
	    if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy initial Size Dev2Host!\n");
		else m_initial_size = temp_initialsize;

	float temp_initialspacing[3] = {0.0, 0.0,0.0};
	cudaStatus = hipMemcpy( temp_initialspacing, (const float*) dev_initial_spacing, 3*sizeof(float),  hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy initial Spacing Dev2Host!\n");
		else m_initial_spacing = temp_initialspacing;

	float temp_initialorigen[3] = {0.0,0.0,0.0};
	cudaStatus = hipMemcpy( temp_initialorigen, (const float*) dev_initial_origen, 3*sizeof(float),  hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy initial Origen Dev2Host!\n");
		else m_initial_origen = temp_initialorigen;

	float* imageinitialpointer = new float[numberOfPixels_initial];
	cudaStatus = hipMemcpy( imageinitialpointer, (const float*) dev_initial_imagepointer, numberOfPixels_initial*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy initial image pointer Dev2Host!\n");
		else m_initial_imagepointer = imageinitialpointer;

	int temp_simuladasize[3] = {0,0,0};
	cudaStatus = hipMemcpy( temp_simuladasize, (const int*) dev_simulada_size, 3*sizeof(int), hipMemcpyDeviceToHost);
	    if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy simulada Size Dev2Host!\n");
		else m_simulada_size = temp_simuladasize;

	float temp_simuladaspacing[3] = {0.0,0.0,0.0};
	cudaStatus = hipMemcpy( temp_simuladaspacing, (const float*) dev_simulada_spacing, 3*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy simulada Spacing Dev2Host!\n");
		else m_simulada_spacing = temp_simuladaspacing;

	float temp_simuladaorigen[3] = {0.0,0.0,0.0};
	cudaStatus = hipMemcpy( temp_simuladaorigen, (const float*) dev_simulada_origen, 3*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy simulada Origen Dev2Host!\n");
		else m_simulada_origen = temp_simuladaorigen;
*/
//	printf("va a inicializar con pixels \n" );
	//float * temp_imagesimuladapointer;
	 //float * temp_imagesimuladapointer = new float[numberOfPixels_Simulada]; // El m_... no convenci\F3 quiz\E1 porque no esta ba inicializado...

	cudaStatus = hipMemcpy(m_simulada_imagepointer, (const char*) dev_simulada_imagepointer, numberOfPixels_Simulada*sizeof(char),  hipMemcpyDeviceToHost);
	//cudaStatus = hipMemcpy(m_simulada_imagepointer, (const unsigned short*)dev_simulada_imagepointer, numberOfPixels_Simulada*sizeof(unsigned short),  hipMemcpyDeviceToHost);

//		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy simulada image pointer Dev2Host!\n");
//		else  m_parameters->simulada_imagePointer = m_simulada_imagepointer;
	
/*  AQUI VA LA RECUPERACION DE LA IMAGEN ORIGINAL !! RECUERDALO PORQUE ESTO HABRA QUE CAMBIARLO

	printf("va a inicializar con pixels \n" );
	//float * temp_imagesimuladapointer;
	 float * temp_imagesimuladapointer = new float[numberOfPixels_Simulada]; // El m_... no convenci\F3 quiz\E1 porque no esta ba inicializado...
	//cudaStatus = hipMemcpy(temp_imagesimuladapointer, (const float*) dev_simulada_imagepointer, numberOfPixels_Simulada*sizeof(float),  hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(temp_imagesimuladapointer, (const float*)dev_simulada_imagepointer, numberOfPixels_Simulada*sizeof(float),  hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy simulada image pointer Dev2Host!\n");
		else {
			m_simulada_imagepointer = temp_imagesimuladapointer;
			m_parameters->simulada_imagePointer = temp_imagesimuladapointer;
		}


*/


/*	float temp_source[3] = {0.0,0.0,0.0};
	cudaStatus = hipMemcpy( temp_source, (const float*) dev_source, 3*sizeof(float), hipMemcpyDeviceToHost);
	    if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy Source Dev2Host!\n");
		else m_source = temp_source;
*/

/*
	printf("\n");
	printf("initial size: [ %i,%i,%i]\n", m_initial_size[0],m_initial_size[1],m_initial_size[2]);
	printf("initial Spacing : [ %f,%f,%f]\n",m_initial_spacing[0], m_initial_spacing[1], m_initial_spacing[2]);
	printf("initial Origen : [ %f,%f,%f]\n",m_initial_origen[0], m_initial_origen[1], m_initial_origen[2]);
	printf("initial Image Pointer : [ %f,%f,%f, ...]\n",m_initial_imagepointer[0], m_initial_imagepointer[1], m_initial_imagepointer[2]);


	printf("simulada Size : [ %d,%d,%d]\n",m_simulada_size[0], m_simulada_size[1], m_simulada_size[2]);
	printf("simulada Spacing : [ %f,%f,%f]\n",m_simulada_spacing[0], m_simulada_spacing[1], m_simulada_spacing[2]);
	printf("simulada Origen : [ %f,%f,%f]\n",m_simulada_origen[0], m_simulada_origen[1], m_simulada_origen[2]);
	printf("simulada Image Pointer : [ %f,%f,%f, ...]\n",m_simulada_imagepointer[0], m_simulada_imagepointer[1], m_simulada_imagepointer[2]);

	printf("Source : [ %f,%f,%f]\n",m_source[0], m_source[1], m_source[2]);
	printf("\n");
*/

// Liberando memoria !!
	hipFree( (void*) dev_initial_size);				//hipFree( temp_initialsize);
	hipFree( (void*) dev_initial_spacing);			//hipFree( temp_initialspacing);
	hipFree( (void*) dev_initial_origen );			//hipFree( temp_initialorigen);
	hipFree( (void*) dev_initial_imagepointer );	//hipFree( imageinitialpointer);

	hipFree( (void*) dev_i_points );
	hipFree( (void*) dev_f_points );
	hipFree( (void*) dev_elements );

	hipFree( (void*) dev_grid_origen );
	hipFree( (void*) dev_grid_spacing );
	hipFree( (void*) dev_grid_size );

	hipFree( (void*) dev_flags );
	hipFree( (void*) dev_cumsum );
	hipFree( (void*) dev_correspondingElements );

	hipFree( (void*) dev_simulada_size);				//hipFree( temp_simuladasize);
	hipFree( (void*) dev_simulada_spacing);			//hipFree( temp_simuladaspacing);
	hipFree( (void*) dev_simulada_origen);			//hipFree( temp_simuladaorigen);
	hipFree( (void*) dev_simulada_imagepointer);		//hipFree( (void*) temp_imagesimuladapointer);

	//hipFree( numberOfPixels_initial);		hipFree( numberOfPixels_simulada);

//	hipFree( (void*) dev_source);				//hipFree( temp_source);
	hipDeviceReset();

	// hipFree( kernel_projection );

// Time !!
/*	hipEventRecord(stop,0);
	//hipEventSynchronize( stop);
	hipEventElapsedTime( &time, start, stop);
	printf( "Time: %f ms.\n", time);

	printf("\n");
	printf("Sale de cuda\n");
	printf("\n");
	*/
}
